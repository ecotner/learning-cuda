#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include <cstdio>
#include <ctime>

// global counter to count points that fall into circle
__device__ int dnum = 0;

__global__ void countPoints(float* xs, float* ys) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float x = xs[idx];
    float y = ys[idx];
    int n = (x*x + y*y < 1.0f) ? 1 : 0;
    // int n = 1;
    atomicAdd(&dnum, n);
}

int main() {
    // number of points that we're going to generate
    const int count = 512*512; // 262144
    const int size = count * sizeof(float);
    // status/error variables?
    hipError_t cudaStatus;
    hiprandStatus_t hiprandStatus;
    // random number generator
    hiprandGenerator_t gen;

    // initialize random number generator
    hiprandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

    // allocate memory on device for (x,y) coordinates of points
    float *x, *y;
    cudaStatus = hipMalloc(&x, size);
    cudaStatus = hipMalloc(&y, size);

    // generate a bunch of random numbers for x and y between [0, 1]
    hiprandStatus = hiprandGenerateUniform(gen, x, count);
    hiprandStatus = hiprandGenerateUniform(gen, y, count);

    // count the points that fall inside the circle
    countPoints<<<512,512>>>(x, y);

    // copy the result back to host
    int hnum;
    // why does function declaration say `dnum` should be a pointer, but
    // it doesn't work when I pass `&dnum`? does `__device__` implicitly
    // declare a pointer?
    hipMemcpyFromSymbol(&hnum, HIP_SYMBOL(dnum), sizeof(int));
    hipFree(x);
    hipFree(y);

    // print result
    float pi = 4.0f * ((float)hnum / (float)count);
    printf("pi is approximately %f\n", pi);
    return cudaStatus | hiprandStatus;
    // compile with `nvcc -lcurand monte_carlo_pi.cu` to include curand lib
}