#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <cmath>
#include <ctime>

const int chunkCount = 1 << 20; // 2^20 ~ 10^6
const int totalCount = chunkCount << 3; // 2^23 ~ 8*10^6

// add two numbers together and take error function of result, store in array
__global__ void kernel(float* a, float* b, float* c) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < chunkCount)
        c[tid] = erff(a[tid] + b[tid]);
}

int main() {
    // get device properties
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    // if device overlap is not possible, we can't do this demo
    if (!prop.deviceOverlap) {
        printf("Device does not have GPU_OVERLAP\n");
        exit(0);
    }


    // initialize events
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // initialize streams
    // *** note that we have TWO streams now ***
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipStream_t stream2;
    hipStreamCreate(&stream2);

    // declare host/device arrays
    // since we have two streams (and don't want thread collisions),
    // we'll need two copies of each device array
    float *ha, *hb, *hc, *d1a, *d1b, *d1c, *d2a, *d2b, *d2c;

    // allocate memory
    const int totalSize = totalCount * sizeof(float);
    const int chunkSize = chunkCount * sizeof(float);
    hipMalloc(&d1a, chunkSize);
    hipMalloc(&d1b, chunkSize);
    hipMalloc(&d1c, chunkSize);
    hipMalloc(&d2a, chunkSize);
    hipMalloc(&d2b, chunkSize);
    hipMalloc(&d2c, chunkSize);
    // use pinned memory here for faster data transfer.
    // we will be doing multiple transfers because of the
    // chunking, so it will be worth the allocation overhead.
    hipHostAlloc(&ha, totalSize, hipHostMallocDefault);
    hipHostAlloc(&hb, totalSize, hipHostMallocDefault);
    hipHostAlloc(&hc, totalSize, hipHostMallocDefault);

    // fill a and b with some random values
    srand((unsigned) time(0));
    for (int i=0; i < totalCount; i++) {
        // generate random numbers between [0,1]
        ha[i] = rand()/RAND_MAX;
        hb[i] = rand()/RAND_MAX;
    }

    // start recording event stream
    hipEventRecord(start, stream1);
    // split data into chunks and iterate over two chunks at a time (interleaving the two streams)
    for (int i=0; i<totalCount; i+=2*chunkCount) {
        int i1 = i;
        int i2 = i + chunkCount;
        hipMemcpyAsync(d1a, ha+i1, chunkSize, hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d2a, ha+i2, chunkSize, hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(d1b, ha+i1, chunkSize, hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d2b, ha+i2, chunkSize, hipMemcpyHostToDevice, stream2);
        kernel<<<chunkCount/64,64,0,stream1>>>(d1a, d1b, d1c);
        kernel<<<chunkCount/64,64,0,stream2>>>(d2a, d2b, d2c);
        hipMemcpyAsync(hc+i1, d1c, chunkSize, hipMemcpyDeviceToHost);
        hipMemcpyAsync(hc+i2, d2c, chunkSize, hipMemcpyDeviceToHost);
    }
    // wait until streams reach here, record end event
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipEventRecord(end);
    hipEventSynchronize(end);
    // get total elapsed time
    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);

    // print results
    printf("This took %f ms\n", elapsed);

    // free memory
    hipHostFree(ha);
    hipHostFree(hb);
    hipHostFree(hc);
    hipFree(d1a);
    hipFree(d1a);
    hipFree(d1b);
    hipFree(d2c);
    hipFree(d2b);
    hipFree(d2c);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}

// executing this on my device results in:
// This took 8.091616 ms

// compare this with the single-stream application, which had an
// elapsed time of 8.373248 ms (3.4% reduction). it isn't a super
// crazy speedup, but it is a speedup nonetheless thanks to streams