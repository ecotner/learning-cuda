#include "hip/hip_runtime.h"
/*
this is the same file as the atomic sum computation in the previous chapter,
only now we will be profiling it using CUDA events.
*/
#include "hip/hip_runtime.h"
#include ""
// header file where `atomicAdd` is defined (or maybe not?)
#include "sm_60_atomic_functions.h"
#include <cstdio>

// declare some global memory on the device
__device__ int dSum = 0;

__global__ void sum(int* d) {
    int tid = threadIdx.x;
    // this would be a naiive way to increment the value, but results in threads writing
    // multiple different values to the same memory location, causing races
    // dSum += d[tid]; 
    // this blocks all other threads so that only one thread at a time may modify the `dSum` variable
    atomicAdd(&dSum, d[tid]);
}

int main() {
    // initialize a vector of integers
    const int count = 128;
    const int size = count * sizeof(int);
    int h[count];
    for (int i=0; i<count; i++) {
        h[i] = i+1;
    }


    // copy that vector over to the device
    int* d;
    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    // initialize CUDA event
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // start recording the event, run the kernel, then record another event
    hipEventRecord(start);
    sum<<<1,count>>>(d);
    hipEventRecord(end);
    // call this to make sure the CPU/GPU are in sync
    hipEventSynchronize(end);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);

    // read result back into host memory and print
    int hSum;
    hipMemcpyFromSymbol(&hSum, HIP_SYMBOL(dSum), sizeof(int));
    printf("The sum of numbers from 1 to %d is %d\n", count, hSum);
    printf("And it took %f msec\n", elapsed);
    hipFree(d);
}